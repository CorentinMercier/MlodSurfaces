#include "hip/hip_runtime.h"
// --------------------------------------------------------------------------
// Source code provided FOR REVIEW ONLY, as part of the submission entitled
// "Moving Level-of-Detail Surfaces".
//
// A proper version of this code will be released if the paper is accepted
// with the proper licence, documentation and bug fix.
// Currently, this material has to be considered confidential and shall not
// be used outside the review process.
//
// All right reserved. The Authors
// --------------------------------------------------------------------------

#include "Fastapss.h"
#include "GPUUtils.cu"
#include "GlobalAPSS.cu"

Fastapss::Fastapss(APSSOctree *apssOctree)
{
	m_apssOctree = apssOctree;
	std::cout << "FastAPSS method used" << std::endl;
}

Fastapss::Fastapss(APSSOctree *apssOctree, const Kernel &kernel, unsigned int nbOfVectors):Fastapss(apssOctree)
{
	copyToGPU(kernel, nbOfVectors);
}


//////////////////////////////////////////////////////////////////////////
///
/// GPU structs declarations and copies
///
//////////////////////////////////////////////////////////////////////////

struct apssNodeStatsGPUSoA{
	unsigned int m_nbOfLeaves;
	float3 * s_ai_pi = NULL;
	float3* s_ai_ni = NULL;
	float * s_ai_pi_ni = NULL;
	float * s_ai_pi_pi = NULL;
	float * s_ai = NULL;

	__host__ void init(unsigned int nbOfLeaves)
	{
		m_nbOfLeaves = nbOfLeaves;
		hipMallocManaged(&s_ai_pi, m_nbOfLeaves * sizeof (float3));
		hipMallocManaged(&s_ai_ni, m_nbOfLeaves * sizeof (float3));
		hipMallocManaged(&s_ai_pi_ni, m_nbOfLeaves * sizeof (float));
		hipMallocManaged(&s_ai_pi_pi, m_nbOfLeaves * sizeof (float));
		hipMallocManaged(&s_ai, m_nbOfLeaves * sizeof (float));
	}
	__host__ void erase()
	{
		if (s_ai_pi){ gpuErrchk(hipFree(s_ai_pi)); s_ai_pi = NULL;}
		if (s_ai_ni){ gpuErrchk(hipFree(s_ai_ni)); s_ai_ni = NULL;}
		if (s_ai_pi_ni){ gpuErrchk(hipFree(s_ai_pi_ni)); s_ai_pi_ni = NULL;}
		if (s_ai_pi_pi){ gpuErrchk(hipFree(s_ai_pi_pi)); s_ai_pi_pi = NULL;}
		if (s_ai){ gpuErrchk(hipFree(s_ai)); s_ai = NULL;}
	}
	__host__ void copy(std::vector<apssNodeStats> const & a)
	{
		init(a.size());
		for (unsigned int i=0; i<a.size(); i++)
		{
			s_ai_ni[i] = make_float3(a[i].s_ai_ni.x, a[i].s_ai_ni.y, a[i].s_ai_ni.z);
			s_ai_pi[i] = make_float3(a[i].s_ai_pi.x, a[i].s_ai_pi.y, a[i].s_ai_pi.z);
			s_ai_pi_ni[i] = a[i].s_ai_pi_ni;
			s_ai_pi_pi[i] = a[i].s_ai_pi_pi;
			s_ai[i] = a[i].s_ai;
		}
	}
};

__device__ void func(float x, float & result)
{
#ifdef EXPFUNC
	if (x>=1){
		result = 1;
	}
	else
		result = __expf(__fdiv_rn(-(__expf(__fdiv_rn(1, x-1))), x*x));
#else
	result = (1 - cos(3.14159265 * x))/2.0;
#endif
}

void copy(octreeNode const * a, octreeNodeGPU2 * b, unsigned int numberOfNodes)
{
	std::vector<const octreeNode*> nodes;
	std::vector<relations> related;
	nodes.resize(numberOfNodes, nullptr);
	related.resize(numberOfNodes);
	a->getRelations(nodes, related);
	for(unsigned int i=0; i<numberOfNodes; i++)
	{
		b[i].depth = nodes[i]->depth;
		b[i].radius = float(nodes[i]->getBoundingBox().radius());
		b[i].center = glmToFloat3(nodes[i]->getBoundingBox().center());
		b[i].s_ai_pi = glmToFloat3(nodes[i]->nodeapssNodeStats.s_ai_pi);
		b[i].s_ai_ni = glmToFloat3(nodes[i]->nodeapssNodeStats.s_ai_ni);
		b[i].s_ai_pi_ni = nodes[i]->nodeapssNodeStats.s_ai_pi_ni;
		b[i].s_ai_pi_pi = nodes[i]->nodeapssNodeStats.s_ai_pi_pi;
		b[i].s_ai = nodes[i]->nodeapssNodeStats.s_ai;
		b[i].indicesSize = nodes[i]->indices.size();
		if (b[i].indicesSize != 0)
		{
			hipMallocManaged(&b[i].indices, b[i].indicesSize * sizeof (unsigned int));
			for (unsigned int j=0; j<b[i].indicesSize; j++)
			{
				b[i].indices[j] = nodes[i]->indices[j];
			}
		}
		b[i].numberOfChildren = nodes[i]->numberOfChildren;
		b[i].father = related[i].father;
		b[i].nextBrother = related[i].nextBrother;
		b[i].firstChild = related[i].firstChild;
	}
}


//////////////////////////////////////////////////////////////////////////
///
/// GPU Version of FastAPSS -> Non recursive version (use of while) with Array of Structs
///
//////////////////////////////////////////////////////////////////////////


__device__
#ifdef SHARED_MEM
void approximateAPSSNonRecursive( float3 const & q , KernelGPU const * kernel , unsigned int const & minimal_depth , float const & scalingProtectionSphere ,
								  apssNodeStatsGPUSoA const & leavesapssNodeStats,
							 const octreeNodeGPU2 *__restrict  node,
							 apssStatsGPU & treeStats, const octreeNodeGPU2 * nodeInSharedMemory, const unsigned sharedMemorySize)
#else
void approximateAPSSNonRecursive( float3 const & q , KernelGPU const * kernel , unsigned int const & minimal_depth , float const & scalingProtectionSphere ,
								  apssNodeStatsGPUSoA const & leavesapssNodeStats,
							 const octreeNodeGPU2 *__restrict  node,
							 apssStatsGPU & treeStats )
#endif
{
	apssStatsGPU depthStats[MAX_DEPTH];
	int currentChild[MAX_DEPTH - 1];
	for (unsigned int i=0; i<MAX_DEPTH - 1; i++)
	{
		depthStats[i].init();
		currentChild[i] = -1;
	}
	depthStats[MAX_DEPTH - 1].init();
	bool not_finished = true;
#ifdef SHARED_MEM
	octreeNodeGPU2 currentNode = nodeInSharedMemory[0];
#else
	octreeNodeGPU2 currentNode = node[0];
#endif
	while(not_finished)
	{
		if( currentNode.indicesSize > 0 ) { //Leaf
#pragma unroll 4
			for( int lIt = 0 ; lIt < currentNode.indicesSize ; ++lIt ) {
				unsigned int leafIdx = currentNode.indices[lIt];
				float w = kernel->w(leavesapssNodeStats.s_ai_pi[leafIdx] / leavesapssNodeStats.s_ai[leafIdx], q);

				depthStats[currentNode.depth].s_ai_wi = __fmaf_rn(w, leavesapssNodeStats.s_ai[leafIdx], depthStats[currentNode.depth].s_ai_wi);
				depthStats[currentNode.depth].s_ai_wi_ni = depthStats[currentNode.depth].s_ai_wi_ni + w * leavesapssNodeStats.s_ai_ni[leafIdx];
				depthStats[currentNode.depth].s_ai_wi_pi = depthStats[currentNode.depth].s_ai_wi_pi + w * leavesapssNodeStats.s_ai_pi[leafIdx];
				depthStats[currentNode.depth].s_ai_wi_pi_ni = __fmaf_rn(w, leavesapssNodeStats.s_ai_pi_ni[leafIdx], depthStats[currentNode.depth].s_ai_wi_pi_ni);
				depthStats[currentNode.depth].s_ai_wi_pi_pi = __fmaf_rn(w, leavesapssNodeStats.s_ai_pi_pi[leafIdx], depthStats[currentNode.depth].s_ai_wi_pi_pi);
			}
			int nextNode = currentNode.father;
			if (nextNode != -1)
			{
#ifdef SHARED_MEM
				if (nextNode < sharedMemorySize)
					currentNode = nodeInSharedMemory[nextNode];
				else
#endif
					currentNode = node[nextNode];
			}
			else
				not_finished = false;
		}
		else if (currentNode.depth >= minimal_depth) {
			float3 p = currentNode.s_ai_pi / currentNode.s_ai;
			if( norme(q - currentNode.center)  > scalingProtectionSphere * currentNode.radius )
			{
				float w = kernel->w( p , q );
				depthStats[currentNode.depth].s_ai_wi = __fmaf_rn(w, currentNode.s_ai, depthStats[currentNode.depth].s_ai_wi);
				depthStats[currentNode.depth].s_ai_wi_ni = depthStats[currentNode.depth].s_ai_wi_ni + w * currentNode.s_ai_ni;
				depthStats[currentNode.depth].s_ai_wi_pi = depthStats[currentNode.depth].s_ai_wi_pi + w * currentNode.s_ai_pi;
				depthStats[currentNode.depth].s_ai_wi_pi_ni = __fmaf_rn(w, currentNode.s_ai_pi_ni, depthStats[currentNode.depth].s_ai_wi_pi_ni);
				depthStats[currentNode.depth].s_ai_wi_pi_pi = __fmaf_rn(w, currentNode.s_ai_pi_pi, depthStats[currentNode.depth].s_ai_wi_pi_pi);

				int nextNode = currentNode.father;
				if (nextNode != -1)
				{
#ifdef SHARED_MEM
					if (nextNode < sharedMemorySize)
						currentNode = nodeInSharedMemory[nextNode];
					else
#endif
						currentNode = node[nextNode];
				}
				else
					not_finished = false;
			}
			else {
				// do blending between levels

				if (currentChild[currentNode.depth] != -1) //Computation done for the child currentChild
				{
					apssStatsGPU parentStats;
					float w = kernel->w( p , q ) * node[currentChild[currentNode.depth]].s_ai / currentNode.s_ai ;
					parentStats.init(w, currentNode);

					float distToParent = fabs(norme(q - currentNode.center) - (currentNode.radius * scalingProtectionSphere));

					float distChild = max(norme(q - node[currentChild[currentNode.depth]].center) - (node[currentChild[currentNode.depth]].radius * scalingProtectionSphere),0.f);
					float wChild = distToParent / (distToParent + distChild);
					float valChild;
					func(wChild, valChild);
					float valParent = 1 - valChild;

					depthStats[currentNode.depth+1].times(depthStats[currentNode.depth+1], valChild);
					depthStats[currentNode.depth].add(depthStats[currentNode.depth+1]);
					parentStats.times(parentStats, valParent);
					depthStats[currentNode.depth].add(parentStats);

					//Reinitialize this level
					depthStats[currentNode.depth+1].init();

					int nextNode = node[currentChild[currentNode.depth]].nextBrother;
					if (nextNode != -1)
					{
						currentChild[currentNode.depth] = nextNode;
#ifdef SHARED_MEM
						if (nextNode < sharedMemorySize)
							currentNode = nodeInSharedMemory[nextNode];
						else
#endif
							currentNode = node[nextNode];
					}
					else {
						currentChild[currentNode.depth] = -1;
						nextNode = currentNode.father;
						if (nextNode != -1)
						{
#ifdef SHARED_MEM
							if (nextNode < sharedMemorySize)
								currentNode = nodeInSharedMemory[nextNode];
							else
#endif
								currentNode = node[nextNode];
						}
						else
							not_finished = false;
					}
				}
				else {
					currentChild[currentNode.depth] = currentNode.firstChild;
#ifdef SHARED_MEM
					if (currentNode.firstChild < sharedMemorySize)
						currentNode = nodeInSharedMemory[currentNode.firstChild];
					else
#endif
						currentNode = node[currentNode.firstChild];
				}
			}
		}

		// if not, then just accumulate the wn of the children:
		else
		{
			if (currentChild[currentNode.depth] != -1) //Computation done for the child currentChild
			{
				depthStats[currentNode.depth].add(depthStats[currentNode.depth+1]);
				//Reinitialize this level
				depthStats[currentNode.depth+1].init();
				int nextNode = node[currentChild[currentNode.depth]].nextBrother;
				if (nextNode != -1)
				{
					currentChild[currentNode.depth] = nextNode;
					currentNode = node[nextNode];
				}
				else {
					currentChild[currentNode.depth] = -1;
					nextNode = currentNode.father;
					if (nextNode != -1)
						currentNode = node[nextNode];
					else
						not_finished = false;
				}
			}
			else {
				currentChild[currentNode.depth] = currentNode.firstChild;
				currentNode = node[currentNode.firstChild];
			}
		}

	}
	treeStats.add(depthStats[0]);
}

__device__
#ifdef SHARED_MEM
void oneStepProjection( float3 const & q , float3 & outputPoint , float3 & outputNormal ,
						  const KernelGPU * kernel, unsigned int minimal_depth , float scalingProtectionSphere,
						  apssNodeStatsGPUSoA const & leavesapssNodeStats, const octreeNodeGPU2 *__restrict node, float stepMaxSize, const octreeNodeGPU2 * nodeInSharedMemory, const unsigned sharedMemorySize)
#else
void oneStepProjection( float3 const & q , float3 & outputPoint , float3 & outputNormal ,
								  KernelGPU const * kernel, unsigned int minimal_depth , float scalingProtectionSphere,
						apssNodeStatsGPUSoA const & leavesapssNodeStats, const octreeNodeGPU2 *__restrict node, float stepMaxSize)
#endif
{
	apssStatsGPU treeStats;
	treeStats.init();
#ifdef SHARED_MEM
	approximateAPSSNonRecursive( q , kernel , minimal_depth , scalingProtectionSphere , leavesapssNodeStats, node, treeStats, nodeInSharedMemory, sharedMemorySize);
#else
	approximateAPSSNonRecursive( q , kernel , minimal_depth , scalingProtectionSphere , leavesapssNodeStats, node, treeStats);
#endif

	float u4 = 0.5*( treeStats.s_ai_wi_pi_ni/*/treeStats.s_ai_wi*/ - dot((treeStats.s_ai_wi_pi/*/treeStats.s_ai_wi*/),(treeStats.s_ai_wi_ni/treeStats.s_ai_wi)) ) /
			( treeStats.s_ai_wi_pi_pi/*/treeStats.s_ai_wi*/ - dot((treeStats.s_ai_wi_pi/*/treeStats.s_ai_wi*/),(treeStats.s_ai_wi_pi/treeStats.s_ai_wi)) );
	float3 u123= (treeStats.s_ai_wi_ni - 2*u4*treeStats.s_ai_wi_pi)/treeStats.s_ai_wi;
	float u0= -(dot(treeStats.s_ai_wi_pi,u123) + u4*treeStats.s_ai_wi_pi_pi)/treeStats.s_ai_wi;

	float3 inputPt = q;
	outputPoint = q;

	if( fabs(u4) < 0.000001 ) {
		// then project on a plane (it's a degenerate sphere)
		float3 n = -u123;
		float lambda = ( u0 - dot(outputPoint,n) ) / squaredNorm(n);
		outputPoint = outputPoint + lambda * n;
		outputNormal = treeStats.s_ai_wi_ni;
		outputNormal = normalize(outputNormal);
	}
	else {
		float3 sphere_center = u123/(-2*u4);
		float val = squaredNorm(sphere_center) - u0/u4 > 0.0 ? squaredNorm(sphere_center) - u0/u4 : 0.0;
		float sphere_radius = sqrt( val );

		// projection of the inputpoint onto the sphere:
		float3 pc= outputPoint-sphere_center;
		pc = normalize(pc);
		outputPoint = sphere_center + sphere_radius*pc;

		// compute normal by looking at the gradient there:
		outputNormal = u123 + 2*u4*outputPoint;
		outputNormal = normalize(outputNormal);
	}

	//Invalid points have a normal equals to (0, 0, 0) and their position is kept for debug
	if (isnan(outputPoint.x) || isnan(outputPoint.y) || isnan(outputPoint.z))
	{
		outputPoint = inputPt;
		outputNormal = make_float3(0, 0, 0);
		return;
	}

	float stepSize = norme(outputPoint - inputPt);
	if (stepSize > stepMaxSize)
		outputPoint = inputPt + (stepMaxSize / stepSize) * (outputPoint - inputPt);
}

__global__
void projectCuda(unsigned int n, float3 const * qStart , float3 * outputPoint , float3 * outputNormal , unsigned int n_iterations ,
				 KernelGPU const * kernel, unsigned int minimal_depth, float scalingProtectionSphere,
				 apssNodeStatsGPUSoA const & leavesapssNodeStats, const octreeNodeGPU2 *__restrict node, float stepMaxSize, unsigned blockSize)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

#ifdef SHARED_MEM
	const unsigned sharedMemorySize = 73;
	__shared__ octreeNodeGPU2 nodeInSharedMemory[sharedMemorySize];
	for (int i = threadIdx.x; i < sharedMemorySize; i+=blockSize)
	{
		nodeInSharedMemory[i] = node[i];
	}
	__syncthreads();
	__shared__ KernelGPU k;
	if (threadIdx.x == 0)
		k = *kernel;
	__syncthreads();
	for (int i = threadIdx.x; i < kernel->sigmasSize; i+=blockSize)
	{
		k.sigmas[i] = kernel->sigmas[i];
	}
	__syncthreads();
#endif

	for (int i = index; i < n; i+=stride)
	{
		outputPoint[i] = qStart[i];
		for (unsigned int j=0; j<n_iterations; j++)
		{
#ifdef SHARED_MEM
			oneStepProjection(outputPoint[i], outputPoint[i], outputNormal[i], &k, minimal_depth, scalingProtectionSphere, leavesapssNodeStats, node, stepMaxSize, nodeInSharedMemory, sharedMemorySize);
#else
			oneStepProjection(outputPoint[i], outputPoint[i], outputNormal[i], kernel, minimal_depth, scalingProtectionSphere, leavesapssNodeStats, node, stepMaxSize);
#endif
		}
	}
}

void Fastapss::project(unsigned int nbOfVectors, glm::vec3 *outputPoints, glm::vec3 *outputNormals, unsigned int n_iterations, const Kernel &kernel, float stepMaxSize) const
{
	if (nbOfVectors == 0)
	{
		std::cerr << "No points to project" << std::endl;
		return;
	}
	gpuErrchk( hipPeekAtLastError() );
	int blockSize = 128;
	dim3 numBlocks = computeNbBlocks(nbOfVectors, blockSize);
	if (stepMaxSize < 0)
		stepMaxSize =  m_apssOctree->getBoundingBox().radius();
	projectCuda<<<numBlocks, blockSize>>>(nbOfVectors, m_outPts, m_outPts, m_outNmls, n_iterations, m_kernelCuda, m_minDepth, m_scalingProtectionSphere, *m_apssNodeStatsCudaSoA, m_nodesBis, stepMaxSize, blockSize);
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(outputPoints, m_outPts, nbOfVectors * sizeof (float3), hipMemcpyDeviceToHost));
	gpuErrchk( hipMemcpy(outputNormals, m_outNmls, nbOfVectors * sizeof (float3), hipMemcpyDeviceToHost));
}

void Fastapss::copyToGPU(const Kernel &kernel, unsigned int nbOfVectors)
{
	//Allocate memory on GPU
	gpuErrchk( hipPeekAtLastError() );
	std::cout << "Tree is being copied to GPU memory... " << std::flush;
	m_numberOfNodes = m_apssOctree->get_root()->nbOfNodes();
	std::cout << "Number of nodes : " << m_numberOfNodes << std::endl;
	gpuErrchk( hipMallocManaged(&m_nodesBis, m_numberOfNodes * sizeof (octreeNodeGPU2)));
	gpuErrchk( hipMallocManaged(&m_apssNodeStatsCudaSoA, sizeof(apssNodeStatsGPUSoA)));
	gpuErrchk( hipMallocManaged(&m_kernelCuda, sizeof(KernelGPU)));

	gpuErrchk( hipPeekAtLastError() );
	//Fill GPU memory
	copy(m_apssOctree->get_root(), m_nodesBis, m_numberOfNodes);
	gpuErrchk( hipPeekAtLastError() );
	m_kernelCuda->copy(kernel);
	gpuErrchk( hipPeekAtLastError() );
	m_apssNodeStatsCudaSoA->copy(m_apssOctree->getPointsApssNodeStats());
	std::cout << "    Done" << std::endl;

	gpuErrchk( hipPeekAtLastError() );
	std::cout << "Tree depth: " << m_apssOctree->get_root()->computeDepth() << std::endl;
}

void Fastapss::eraseFromGPU()
{
	gpuErrchk( hipPeekAtLastError() );
	hipFree(m_kernelCuda);
	m_apssNodeStatsCudaSoA->erase();
	hipFree(m_apssNodeStatsCudaSoA);
	gpuErrchk( hipPeekAtLastError() );
	for (unsigned int i=0; i<m_numberOfNodes; i++)
		m_nodesBis[i].erase();
	hipFree(m_nodesBis);
	gpuErrchk( hipPeekAtLastError() );
}

void Fastapss::projectCPU(const glm::vec3 &qStart, glm::vec3 &outputPoint, glm::vec3 &outputNormal, unsigned int n_iterations, const Kernel &kernel, float stepMaxSize) const
{
	m_apssOctree->projectCPU(qStart, outputPoint, outputNormal, n_iterations, true, kernel, stepMaxSize);
}
