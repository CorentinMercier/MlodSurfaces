#include "hip/hip_runtime.h"
// --------------------------------------------------------------------------
// This file is part of the reference implementation for the paper
//    Moving Level-of-Detail Surfaces.
//    C. Mercier, T. Lescoat, P. Roussillon, T. Boubekeur, and J-M. Thiery
//    ACM Transaction On Graphics 2022
//    DOI: 10.1145/3528223.3530151
//
// All rights reserved. Use of this source code is governed by a
// MIT license that can be found in the LICENSE file.
// --------------------------------------------------------------------------

#pragma once

#include "GlobalAPSS.h"
#include "GPUUtils.cu"

/// Structures
///
///

struct apssNodeStatsGPU{
	float3 s_ai_pi , s_ai_ni;
	float s_ai_pi_ni , s_ai_pi_pi;
	float s_ai;

	void init()
	{
	s_ai_pi = make_float3(0.0, 0.0, 0.0);
	s_ai_ni = make_float3(0.0, 0.0, 0.0);
	s_ai_pi_ni = 0;
	s_ai_pi_pi = 0;
	s_ai = 0;
	}

	__host__ void copy(apssNodeStats const & a)
	{
	s_ai = a.s_ai;
	s_ai_ni = make_float3(a.s_ai_ni.x, a.s_ai_ni.y, a.s_ai_ni.z);
	s_ai_pi = make_float3(a.s_ai_pi.x, a.s_ai_pi.y, a.s_ai_pi.z);
	s_ai_pi_ni = a.s_ai_pi_ni;
	s_ai_pi_pi = a.s_ai_pi_pi;
	}
};

struct octreeNodeGPU {
	octreeNodeGPU * children[8];
	unsigned int numberOfChildren;
	unsigned int *indices;
	unsigned int indicesSize;
	unsigned int depth;
	BBOXGPU boundingBox;
	void setBoundingBox( const BBOXGPU & bbox ) { boundingBox = bbox; }
	BBOXGPU const & getBoundingBox() const { return boundingBox; }

	apssNodeStatsGPU nodeapssNodeStats;

	__host__ void erase()
	{
		if (indices != 0)
			gpuErrchk( hipFree(indices));
		for (unsigned int i=0; i<8; i++)
		{
			if (children[i] != NULL)
			hipFree(children[i]);
		}
		if(children) hipFree(children);
	}
	__host__ void copy(octreeNode const * a)
	{
		numberOfChildren = a->numberOfChildren;
		depth = a->depth;
		boundingBox.copy(a->boundingBox);
		indicesSize = a->indices.size();
		if (indicesSize != 0)
			gpuErrchk( hipMallocManaged(&indices, a->indices.size() * sizeof (unsigned int)));
		for (unsigned int i=0; i<a->indices.size(); i++)
			indices[i] = a->indices[i];
		nodeapssNodeStats.copy(a->nodeapssNodeStats);
		gpuErrchk( hipMallocManaged((void**)(&children), 8 * sizeof (octreeNodeGPU*)));
		for (unsigned int i=0; i<8; i++)
		{
			if (a->children[i] != NULL)
			{
				gpuErrchk( hipMallocManaged(&children[i], sizeof (octreeNodeGPU)));
				children[i]->copy(a->children[i]);
			}
			else
			{
				children[i] = NULL;
			}
		}
	}
};

GlobalAPSS::GlobalAPSS(APSSOctree *apssOctree)
{
	m_apssOctree = apssOctree;
	std::cout << "GlobalAPSS method used" << std::endl;
}

GlobalAPSS::GlobalAPSS(APSSOctree *apssOctree, const Kernel &kernel, unsigned int nbOfVectors): GlobalAPSS(apssOctree)
{
	copyToGPU(kernel, nbOfVectors);
}


//////////////////////////////////////////////////////////////////////////
///
/// GPU version 0 -> No octree, running on the GPU
///
//////////////////////////////////////////////////////////////////////////

__device__
void APSS( unsigned int nbOfPoints, float3 const & q , KernelGPU const * kernel,
		   apssNodeStatsGPU const * leavesapssNodeStats, apssStatsGPU & treeStats)
{
	for (unsigned int i=0; i<nbOfPoints; i++)
	{
		float w = kernel->w(leavesapssNodeStats[i].s_ai_pi / leavesapssNodeStats[i].s_ai, q);
		treeStats.s_ai_wi = __fmaf_rn(w, leavesapssNodeStats[i].s_ai, treeStats.s_ai_wi);
		treeStats.s_ai_wi_ni = treeStats.s_ai_wi_ni + w * leavesapssNodeStats[i].s_ai_ni;
		treeStats.s_ai_wi_pi = treeStats.s_ai_wi_pi + w * leavesapssNodeStats[i].s_ai_pi;
		treeStats.s_ai_wi_pi_ni = __fmaf_rn(w, leavesapssNodeStats[i].s_ai_pi_ni, treeStats.s_ai_wi_pi_ni);
		treeStats.s_ai_wi_pi_pi = __fmaf_rn(w, leavesapssNodeStats[i].s_ai_pi_pi, treeStats.s_ai_wi_pi_pi);
	}
}

__device__
void computeSphere(float3 const & q, float3 & outputPoint, float3 & outputNormal, apssStatsGPU const & treeStats, float stepMaxSize)
{
	// fit a sphere to the tree:

	// algebraic sphere: u4.||X||^2 + u123.X + u0 = 0
	// geometric sphere: ||X-C||^2 - r^2 = 0
	// geometric plane:  (X-C).n = 0
	float u4 = 0.5*( treeStats.s_ai_wi_pi_ni/*/treeStats.s_ai_wi*/ - dot((treeStats.s_ai_wi_pi/*/treeStats.s_ai_wi*/),(treeStats.s_ai_wi_ni/treeStats.s_ai_wi)) ) /
			( treeStats.s_ai_wi_pi_pi/*/treeStats.s_ai_wi*/ - dot((treeStats.s_ai_wi_pi/*/treeStats.s_ai_wi*/),(treeStats.s_ai_wi_pi/treeStats.s_ai_wi)) );
	float3 u123= (treeStats.s_ai_wi_ni - 2*u4*treeStats.s_ai_wi_pi)/treeStats.s_ai_wi;
	float u0= -(dot(treeStats.s_ai_wi_pi,u123) + u4*treeStats.s_ai_wi_pi_pi)/treeStats.s_ai_wi;

	outputPoint = q;

	if( fabs(u4) < 0.000000000001 ) {
	// then project on a plane (it's a degenerate sphere)
	float3 n = -u123;
	float lambda = ( u0 - dot(outputPoint,n) ) / squaredNorm(n);
	outputPoint = outputPoint + lambda * n;
	outputNormal = treeStats.s_ai_wi_ni;
	outputNormal = normalize(outputNormal);
	}
	else {
	float3 sphere_center = u123/(-2*u4);
	float val = squaredNorm(sphere_center) - u0/u4 > 0.0 ? squaredNorm(sphere_center) - u0/u4 : 0.0;
	float sphere_radius = sqrt( val );

	// projection of the inputpoint onto the sphere:
	float3 pc= outputPoint-sphere_center;
	pc = normalize(pc);
	outputPoint = sphere_center + sphere_radius*pc;

	// compute normal by looking at the gradient there:
	outputNormal = u123 + 2*u4*outputPoint;
	outputNormal = normalize(outputNormal);
	}
	float stepSize = norme(outputPoint - q);
	if (stepSize > stepMaxSize)
	outputPoint = q + (stepMaxSize / stepSize) * (outputPoint - q);
}

__global__
void launchComputeSphere(unsigned int n, float3 const * q, float3 * outputPoint, float3 * outputNormal, apssStatsGPU const * treeStats, float stepMaxSize)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i+=stride)
	{
	computeSphere(outputPoint[i], outputPoint[i], outputNormal[i], treeStats[i], stepMaxSize);
	}
}

__global__
//__launch_bounds__(MAX_THREADS_PER_BLOCK, MIN_BLOCKS_PER_MP)
void projectCudaWithoutOctree(unsigned int n, unsigned int nbOfNodes, float3 const * outputPoint,
							  KernelGPU const * kernel, apssNodeStatsGPU const * leavesapssNodeStats, apssStatsGPU * treeStats)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i+=stride)
	{
	APSS(nbOfNodes, outputPoint[i], kernel, leavesapssNodeStats, treeStats[i]);
	}
}

__global__
void initTreeStats(unsigned int n, apssStatsGPU * treeStats)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i+=stride)
	treeStats[i].init();
}

void GlobalAPSS::project(unsigned int nbOfVectors, glm::vec3 * outputPoints , glm::vec3 * outputNormals , unsigned int n_iterations ,
						 Kernel const & kernel, float stepMaxSize) const
{
	if (nbOfVectors == 0)
	{
		std::cerr << "No points to project" << std::endl;
		return;
	}
	gpuErrchk( hipProfilerStart() );
	int blockSize = 128;
	dim3 numBlocks = computeNbBlocks(nbOfVectors, blockSize);

	gpuErrchk( hipPeekAtLastError() );

	//Init treeStats
	initTreeStats<<<numBlocks, blockSize>>>(nbOfVectors, m_treeStats);
	gpuErrchk( hipDeviceSynchronize());
	//Main loop, treeStats filling
	gpuErrchk( hipPeekAtLastError() );
	projectCudaWithoutOctree<<<numBlocks, blockSize>>>(nbOfVectors, m_apssOctree->getNbOfLeaves(), m_outPts, m_kernelCuda, m_apssNodeStatsCuda, m_treeStats);
	gpuErrchk( hipDeviceSynchronize());
	gpuErrchk( hipPeekAtLastError() );
	//Sphere computation
	if (stepMaxSize < 0)
		stepMaxSize =  m_apssOctree->getBoundingBox().radius();
	launchComputeSphere<<<numBlocks, blockSize>>>(nbOfVectors, m_outPts, m_outPts, m_outNmls, m_treeStats, stepMaxSize);
	gpuErrchk( hipDeviceSynchronize());
	gpuErrchk( hipPeekAtLastError() );

	gpuErrchk( hipMemcpy(outputPoints, m_outPts, nbOfVectors * sizeof (float3), hipMemcpyDeviceToHost));
	gpuErrchk( hipMemcpy(outputNormals, m_outNmls, nbOfVectors * sizeof (float3), hipMemcpyDeviceToHost));
	hipProfilerStop();
}

void GlobalAPSS::copyToGPU(const Kernel &kernel, unsigned int nbOfVectors)
{
	//Allocate memory on GPU
	gpuErrchk( hipPeekAtLastError() );
	std::cout << "Memory is being copied to GPU memory... " << std::flush;
	m_numberOfNodes = m_apssOctree->get_root()->nbOfNodes();
	std::cout << "Number of nodes : " << m_numberOfNodes << std::endl;
	unsigned pointsapssNodeStatsSize = m_apssOctree->getPointsApssNodeStats().size();
	gpuErrchk( hipMallocManaged(&m_apssNodeStatsCuda, pointsapssNodeStatsSize * sizeof(apssNodeStatsGPU)));
	gpuErrchk( hipMallocManaged(&m_kernelCuda, sizeof(KernelGPU)));

	gpuErrchk( hipPeekAtLastError() );
	//Fill GPU memory
	m_kernelCuda->copy(kernel);
	gpuErrchk( hipPeekAtLastError() );
	std::cout << pointsapssNodeStatsSize << std::endl;
	for (unsigned int i=0; i<pointsapssNodeStatsSize; i++)
	m_apssNodeStatsCuda[i].copy(m_apssOctree->getPointsApssNodeStats()[i]);
	std::cout << "    Done" << std::endl;

	gpuErrchk( hipPeekAtLastError() );
}

void GlobalAPSS::eraseFromGPU()
{
	gpuErrchk(hipFree(m_apssNodeStatsCuda));
	gpuErrchk(hipFree(m_kernelCuda));
	gpuErrchk(hipFree(m_treeStats));
}

void GlobalAPSS::copyPointsToGPU(unsigned int nbOfVectors, glm::vec3 *outputPoints)
{
	APSS::copyPointsToGPU(nbOfVectors, outputPoints);
	gpuErrchk(hipMallocManaged(&m_treeStats, nbOfVectors * sizeof (apssStatsGPU)));
}

void GlobalAPSS::erasePointsFromGPU()
{
	APSS::erasePointsFromGPU();
	if (m_treeStats)
	{
	gpuErrchk(hipFree(m_treeStats));
	m_treeStats = NULL;
	}
}

void GlobalAPSS::projectCPU(const glm::vec3 &qStart, glm::vec3 &outputPoint, glm::vec3 &outputNormal, unsigned int n_iterations, const Kernel &kernel, float stepMaxSize) const
{
	m_apssOctree->projectCPU(qStart, outputPoint, outputNormal, n_iterations, false, kernel, stepMaxSize);
}

void GlobalAPSS::cpuSphere(glm::vec3 const & qStart, float &u0, glm::vec3 &u123, float &u4, Kernel const & kernel) const
{
	m_apssOctree->cpuSphere(qStart, u0, u123, u4, kernel);
}
