#include "hip/hip_runtime.h"
// --------------------------------------------------------------------------
// This file is part of the reference implementation for the paper
//    Moving Level-of-Detail Surfaces.
//    C. Mercier, T. Lescoat, P. Roussillon, T. Boubekeur, and J-M. Thiery
//    ACM Transaction On Graphics 2022
//    DOI: 10.1145/3528223.3530151
//
// All rights reserved. Use of this source code is governed by a
// MIT license that can be found in the LICENSE file.
// --------------------------------------------------------------------------

#include "Fastapss.h"
#include "GPUUtils.cu"
#include "GlobalAPSS.cu"

Fastapss::Fastapss(APSSOctree *apssOctree)
{
	m_apssOctree = apssOctree;
	std::cout << "FastAPSS method used" << std::endl;
}

Fastapss::Fastapss(APSSOctree *apssOctree, const Kernel &kernel, unsigned int nbOfVectors):Fastapss(apssOctree)
{
	copyToGPU(kernel, nbOfVectors);
}


//////////////////////////////////////////////////////////////////////////
///
/// GPU structs declarations and copies
///
//////////////////////////////////////////////////////////////////////////

struct apssNodeStatsGPUSoA
{
	unsigned int m_nbOfLeaves = 0;
	float3 * s_ai_pi = nullptr;
	float3* s_ai_ni = nullptr;
	float * s_ai_pi_ni = nullptr;
	float * s_ai_pi_pi = nullptr;
	float * s_ai = nullptr;

	__host__ void init(unsigned int nbOfLeaves)
	{
		m_nbOfLeaves = nbOfLeaves;
		gpuErrchk( hipMalloc(&s_ai_pi, m_nbOfLeaves * sizeof (float3)) );
		gpuErrchk( hipMalloc(&s_ai_ni, m_nbOfLeaves * sizeof (float3)) );
		gpuErrchk( hipMalloc(&s_ai_pi_ni, m_nbOfLeaves * sizeof (float)) );
		gpuErrchk( hipMalloc(&s_ai_pi_pi, m_nbOfLeaves * sizeof (float)) );
		gpuErrchk( hipMalloc(&s_ai, m_nbOfLeaves * sizeof (float)) );
	}
	__host__ void erase()
	{
		if (s_ai_pi)    { gpuErrchk(hipFree(s_ai_pi));    s_ai_pi = nullptr; }
		if (s_ai_ni)    { gpuErrchk(hipFree(s_ai_ni));    s_ai_ni = nullptr; }
		if (s_ai_pi_ni) { gpuErrchk(hipFree(s_ai_pi_ni)); s_ai_pi_ni = nullptr; }
		if (s_ai_pi_pi) { gpuErrchk(hipFree(s_ai_pi_pi)); s_ai_pi_pi = nullptr; }
		if (s_ai)       { gpuErrchk(hipFree(s_ai));       s_ai = nullptr; }
	}
	__host__ void copy(std::vector<apssNodeStats> const & a)
	{
		init(a.size());
		std::unique_ptr<float3[]> h_ai_pi    = std::make_unique<float3[]>(size_t(m_nbOfLeaves));
		std::unique_ptr<float3[]> h_ai_ni    = std::make_unique<float3[]>(size_t(m_nbOfLeaves));
		std::unique_ptr<float[]>  h_ai_pi_ni = std::make_unique<float[]>(size_t(m_nbOfLeaves));
		std::unique_ptr<float[]>  h_ai_pi_pi = std::make_unique<float[]>(size_t(m_nbOfLeaves));
		std::unique_ptr<float[]>  h_ai       = std::make_unique<float[]>(size_t(m_nbOfLeaves));
		for(unsigned int i = 0; i < a.size(); i++)
		{
			h_ai_ni[i]    = make_float3(a[i].s_ai_ni.x, a[i].s_ai_ni.y, a[i].s_ai_ni.z);
			h_ai_pi[i]    = make_float3(a[i].s_ai_pi.x, a[i].s_ai_pi.y, a[i].s_ai_pi.z);
			h_ai_pi_ni[i] = a[i].s_ai_pi_ni;
			h_ai_pi_pi[i] = a[i].s_ai_pi_pi;
			h_ai[i]       = a[i].s_ai;
		}
		gpuErrchk( hipMemcpy(s_ai_ni,    h_ai_ni.get(),    m_nbOfLeaves * sizeof(float3), hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(s_ai_pi,    h_ai_pi.get(),    m_nbOfLeaves * sizeof(float3), hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(s_ai_pi_ni, h_ai_pi_ni.get(), m_nbOfLeaves * sizeof(float), hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(s_ai_pi_pi, h_ai_pi_pi.get(), m_nbOfLeaves * sizeof(float), hipMemcpyHostToDevice) );
		gpuErrchk( hipMemcpy(s_ai,       h_ai.get(),       m_nbOfLeaves * sizeof(float), hipMemcpyHostToDevice) );
	}
};

__device__ void func(float x, float & result)
{
#ifdef EXPFUNC
	if (x>=1){
		result = 1;
	}
	else
		result = __expf(__fdiv_rn(-(__expf(__fdiv_rn(1, x-1))), x*x));
#else
	result = (1 - cos(3.14159265 * x))/2.0;
#endif
}

void copy(octreeNode const * a, octreeNodeGPU2 * b, unsigned int numberOfNodes)
{
	std::vector<const octreeNode*> nodes;
	std::vector<relations> related;
	nodes.resize(numberOfNodes, nullptr);
	related.resize(numberOfNodes);
	a->getRelations(nodes, related);

	uint32_t num_indices_to_copy = 0;
	std::vector<unsigned int> whose_indices_to_copy;

	for(unsigned int i=0; i<numberOfNodes; i++)
	{
		b[i].depth = nodes[i]->depth;
		b[i].radius = float(nodes[i]->getBoundingBox().radius());
		b[i].center = glmToFloat3(nodes[i]->getBoundingBox().center());
		b[i].s_ai_pi = glmToFloat3(nodes[i]->nodeapssNodeStats.s_ai_pi);
		b[i].s_ai_ni = glmToFloat3(nodes[i]->nodeapssNodeStats.s_ai_ni);
		b[i].s_ai_pi_ni = nodes[i]->nodeapssNodeStats.s_ai_pi_ni;
		b[i].s_ai_pi_pi = nodes[i]->nodeapssNodeStats.s_ai_pi_pi;
		b[i].s_ai = nodes[i]->nodeapssNodeStats.s_ai;
		b[i].indicesSize = nodes[i]->indices.size();
		if (b[i].indicesSize != 0)
		{
			whose_indices_to_copy.push_back(i);
			num_indices_to_copy += b[i].indicesSize;
		}
		b[i].numberOfChildren = nodes[i]->numberOfChildren;
		b[i].father = related[i].father;
		b[i].nextBrother = related[i].nextBrother;
		b[i].firstChild = related[i].firstChild;
	}

	if(num_indices_to_copy > 0)
	{
		unsigned int* indices = nullptr;
		gpuErrchk(hipMalloc(&indices, num_indices_to_copy * sizeof(unsigned int)));
		uint32_t offset = 0;
		for (unsigned int i : whose_indices_to_copy)
		{
			b[i].indices = indices + offset;
			offset += b[i].indicesSize;
		}

		std::unique_ptr<unsigned int[]> idxCPU = std::make_unique<unsigned int[]>(num_indices_to_copy * sizeof(unsigned int));
		unsigned int* idx = idxCPU.get();
		for(unsigned int i : whose_indices_to_copy)
		{
			for (unsigned int j = 0; j < b[i].indicesSize; j++)
				*(idx++) = nodes[i]->indices[j];
		}
		gpuErrchk( hipMemcpy(indices, idxCPU.get(), num_indices_to_copy * sizeof(unsigned int), hipMemcpyHostToDevice) );
	}
}


//////////////////////////////////////////////////////////////////////////
///
/// GPU Version of FastAPSS -> Non recursive version (use of while) with Array of Structs
///
//////////////////////////////////////////////////////////////////////////


__device__
#ifdef SHARED_MEM
void approximateAPSSNonRecursive( float3 const & q , KernelGPU const * kernel , unsigned int const & minimal_depth , float const & scalingProtectionSphere ,
								  apssNodeStatsGPUSoA const & leavesapssNodeStats,
							 const octreeNodeGPU2 *__restrict  node,
							 apssStatsGPU & treeStats, const octreeNodeGPU2 * nodeInSharedMemory, const unsigned sharedMemorySize)
#else
void approximateAPSSNonRecursive( float3 const & q , KernelGPU const * kernel , unsigned int const & minimal_depth , float const & scalingProtectionSphere ,
								  apssNodeStatsGPUSoA const & leavesapssNodeStats,
							 const octreeNodeGPU2 *__restrict  node,
							 apssStatsGPU & treeStats )
#endif
{
	apssStatsGPU depthStats[MAX_DEPTH];
	int currentChild[MAX_DEPTH - 1];
	for (unsigned int i=0; i<MAX_DEPTH - 1; i++)
	{
		depthStats[i].init();
		currentChild[i] = -1;
	}
	depthStats[MAX_DEPTH - 1].init();
	bool not_finished = true;
#ifdef SHARED_MEM
	octreeNodeGPU2 currentNode = nodeInSharedMemory[0];
#else
	octreeNodeGPU2 currentNode = node[0];
#endif
	while(not_finished)
	{
		if( currentNode.indicesSize > 0 ) { //Leaf
#pragma unroll 4
			for( int lIt = 0 ; lIt < currentNode.indicesSize ; ++lIt ) {
				unsigned int leafIdx = currentNode.indices[lIt];
				float w = kernel->w(leavesapssNodeStats.s_ai_pi[leafIdx] / leavesapssNodeStats.s_ai[leafIdx], q);

				depthStats[currentNode.depth].s_ai_wi = __fmaf_rn(w, leavesapssNodeStats.s_ai[leafIdx], depthStats[currentNode.depth].s_ai_wi);
				depthStats[currentNode.depth].s_ai_wi_ni = depthStats[currentNode.depth].s_ai_wi_ni + w * leavesapssNodeStats.s_ai_ni[leafIdx];
				depthStats[currentNode.depth].s_ai_wi_pi = depthStats[currentNode.depth].s_ai_wi_pi + w * leavesapssNodeStats.s_ai_pi[leafIdx];
				depthStats[currentNode.depth].s_ai_wi_pi_ni = __fmaf_rn(w, leavesapssNodeStats.s_ai_pi_ni[leafIdx], depthStats[currentNode.depth].s_ai_wi_pi_ni);
				depthStats[currentNode.depth].s_ai_wi_pi_pi = __fmaf_rn(w, leavesapssNodeStats.s_ai_pi_pi[leafIdx], depthStats[currentNode.depth].s_ai_wi_pi_pi);
			}
			int nextNode = currentNode.father;
			if (nextNode != -1)
			{
#ifdef SHARED_MEM
				if (nextNode < sharedMemorySize)
					currentNode = nodeInSharedMemory[nextNode];
				else
#endif
					currentNode = node[nextNode];
			}
			else
				not_finished = false;
		}
		else if (currentNode.depth >= minimal_depth) {
			float3 p = currentNode.s_ai_pi / currentNode.s_ai;
			if( norme(q - currentNode.center)  > scalingProtectionSphere * currentNode.radius )
			{
				float w = kernel->w( p , q );
				depthStats[currentNode.depth].s_ai_wi = __fmaf_rn(w, currentNode.s_ai, depthStats[currentNode.depth].s_ai_wi);
				depthStats[currentNode.depth].s_ai_wi_ni = depthStats[currentNode.depth].s_ai_wi_ni + w * currentNode.s_ai_ni;
				depthStats[currentNode.depth].s_ai_wi_pi = depthStats[currentNode.depth].s_ai_wi_pi + w * currentNode.s_ai_pi;
				depthStats[currentNode.depth].s_ai_wi_pi_ni = __fmaf_rn(w, currentNode.s_ai_pi_ni, depthStats[currentNode.depth].s_ai_wi_pi_ni);
				depthStats[currentNode.depth].s_ai_wi_pi_pi = __fmaf_rn(w, currentNode.s_ai_pi_pi, depthStats[currentNode.depth].s_ai_wi_pi_pi);

				int nextNode = currentNode.father;
				if (nextNode != -1)
				{
#ifdef SHARED_MEM
					if (nextNode < sharedMemorySize)
						currentNode = nodeInSharedMemory[nextNode];
					else
#endif
						currentNode = node[nextNode];
				}
				else
					not_finished = false;
			}
			else {
				// do blending between levels

				if (currentChild[currentNode.depth] != -1) //Computation done for the child currentChild
				{
					apssStatsGPU parentStats;
					float w = kernel->w( p , q ) * node[currentChild[currentNode.depth]].s_ai / currentNode.s_ai ;
					parentStats.init(w, currentNode);

					float distToParent = fabs(norme(q - currentNode.center) - (currentNode.radius * scalingProtectionSphere));

					float distChild = max(norme(q - node[currentChild[currentNode.depth]].center) - (node[currentChild[currentNode.depth]].radius * scalingProtectionSphere),0.f);
					float wChild = distToParent / (distToParent + distChild);
					float valChild;
					func(wChild, valChild);
					float valParent = 1 - valChild;

					depthStats[currentNode.depth+1].times(depthStats[currentNode.depth+1], valChild);
					depthStats[currentNode.depth].add(depthStats[currentNode.depth+1]);
					parentStats.times(parentStats, valParent);
					depthStats[currentNode.depth].add(parentStats);

					//Reinitialize this level
					depthStats[currentNode.depth+1].init();

					int nextNode = node[currentChild[currentNode.depth]].nextBrother;
					if (nextNode != -1)
					{
						currentChild[currentNode.depth] = nextNode;
#ifdef SHARED_MEM
						if (nextNode < sharedMemorySize)
							currentNode = nodeInSharedMemory[nextNode];
						else
#endif
							currentNode = node[nextNode];
					}
					else {
						currentChild[currentNode.depth] = -1;
						nextNode = currentNode.father;
						if (nextNode != -1)
						{
#ifdef SHARED_MEM
							if (nextNode < sharedMemorySize)
								currentNode = nodeInSharedMemory[nextNode];
							else
#endif
								currentNode = node[nextNode];
						}
						else
							not_finished = false;
					}
				}
				else {
					currentChild[currentNode.depth] = currentNode.firstChild;
#ifdef SHARED_MEM
					if (currentNode.firstChild < sharedMemorySize)
						currentNode = nodeInSharedMemory[currentNode.firstChild];
					else
#endif
						currentNode = node[currentNode.firstChild];
				}
			}
		}

		// if not, then just accumulate the wn of the children:
		else
		{
			if (currentChild[currentNode.depth] != -1) //Computation done for the child currentChild
			{
				depthStats[currentNode.depth].add(depthStats[currentNode.depth+1]);
				//Reinitialize this level
				depthStats[currentNode.depth+1].init();
				int nextNode = node[currentChild[currentNode.depth]].nextBrother;
				if (nextNode != -1)
				{
					currentChild[currentNode.depth] = nextNode;
					currentNode = node[nextNode];
				}
				else {
					currentChild[currentNode.depth] = -1;
					nextNode = currentNode.father;
					if (nextNode != -1)
						currentNode = node[nextNode];
					else
						not_finished = false;
				}
			}
			else {
				currentChild[currentNode.depth] = currentNode.firstChild;
				currentNode = node[currentNode.firstChild];
			}
		}

	}
	treeStats.add(depthStats[0]);
}

__device__
#ifdef SHARED_MEM
void oneStepProjection( float3 const & q , float3 & outputPoint , float3 & outputNormal ,
						  const KernelGPU * kernel, unsigned int minimal_depth , float scalingProtectionSphere,
						  apssNodeStatsGPUSoA const & leavesapssNodeStats, const octreeNodeGPU2 *__restrict node, float stepMaxSize, const octreeNodeGPU2 * nodeInSharedMemory, const unsigned sharedMemorySize)
#else
void oneStepProjection( float3 const & q , float3 & outputPoint , float3 & outputNormal ,
								  KernelGPU const * kernel, unsigned int minimal_depth , float scalingProtectionSphere,
						apssNodeStatsGPUSoA const & leavesapssNodeStats, const octreeNodeGPU2 *__restrict node, float stepMaxSize)
#endif
{
	apssStatsGPU treeStats;
	treeStats.init();
#ifdef SHARED_MEM
	approximateAPSSNonRecursive( q , kernel , minimal_depth , scalingProtectionSphere , leavesapssNodeStats, node, treeStats, nodeInSharedMemory, sharedMemorySize);
#else
	approximateAPSSNonRecursive( q , kernel , minimal_depth , scalingProtectionSphere , leavesapssNodeStats, node, treeStats);
#endif

	float u4 = 0.5*( treeStats.s_ai_wi_pi_ni/*/treeStats.s_ai_wi*/ - dot((treeStats.s_ai_wi_pi/*/treeStats.s_ai_wi*/),(treeStats.s_ai_wi_ni/treeStats.s_ai_wi)) ) /
			( treeStats.s_ai_wi_pi_pi/*/treeStats.s_ai_wi*/ - dot((treeStats.s_ai_wi_pi/*/treeStats.s_ai_wi*/),(treeStats.s_ai_wi_pi/treeStats.s_ai_wi)) );
	float3 u123= (treeStats.s_ai_wi_ni - 2*u4*treeStats.s_ai_wi_pi)/treeStats.s_ai_wi;
	float u0= -(dot(treeStats.s_ai_wi_pi,u123) + u4*treeStats.s_ai_wi_pi_pi)/treeStats.s_ai_wi;

	float3 inputPt = q;
	outputPoint = q;

	if( fabs(u4) < 0.000001 ) {
		// then project on a plane (it's a degenerate sphere)
		float3 n = -u123;
		float lambda = ( u0 - dot(outputPoint,n) ) / squaredNorm(n);
		outputPoint = outputPoint + lambda * n;
		outputNormal = treeStats.s_ai_wi_ni;
		outputNormal = normalize(outputNormal);
	}
	else {
		float3 sphere_center = u123/(-2*u4);
		float val = squaredNorm(sphere_center) - u0/u4 > 0.0 ? squaredNorm(sphere_center) - u0/u4 : 0.0;
		float sphere_radius = sqrt( val );

		// projection of the inputpoint onto the sphere:
		float3 pc= outputPoint-sphere_center;
		pc = normalize(pc);
		outputPoint = sphere_center + sphere_radius*pc;

		// compute normal by looking at the gradient there:
		outputNormal = u123 + 2*u4*outputPoint;
		outputNormal = normalize(outputNormal);
	}

	//Invalid points have a normal equals to (0, 0, 0) and their position is kept for debug
	if (isnan(outputPoint.x) || isnan(outputPoint.y) || isnan(outputPoint.z))
	{
		outputPoint = inputPt;
		outputNormal = make_float3(0, 0, 0);
		return;
	}

	float stepSize = norme(outputPoint - inputPt);
	if (stepSize > stepMaxSize)
		outputPoint = inputPt + (stepMaxSize / stepSize) * (outputPoint - inputPt);
}

__global__
void projectCuda(unsigned int n, float3 const * qStart , float3 * outputPoint , float3 * outputNormal , unsigned int n_iterations ,
				 KernelGPU const * kernel, unsigned int minimal_depth, float scalingProtectionSphere,
				 apssNodeStatsGPUSoA const & leavesapssNodeStats, const octreeNodeGPU2 *__restrict node, float stepMaxSize, unsigned blockSize)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;

#ifdef SHARED_MEM
	const unsigned sharedMemorySize = 73;
	__shared__ octreeNodeGPU2 nodeInSharedMemory[sharedMemorySize];
	for (int i = threadIdx.x; i < sharedMemorySize; i+=blockSize)
	{
		nodeInSharedMemory[i] = node[i];
	}
	__syncthreads();
	__shared__ KernelGPU k;
	if (threadIdx.x == 0)
		k = *kernel;
	__syncthreads();
	for (int i = threadIdx.x; i < kernel->sigmasSize; i+=blockSize)
	{
		k.sigmas[i] = kernel->sigmas[i];
	}
	__syncthreads();
#endif

	for (int i = index; i < n; i+=stride)
	{
		outputPoint[i] = qStart[i];
		for (unsigned int j=0; j<n_iterations; j++)
		{
#ifdef SHARED_MEM
			oneStepProjection(outputPoint[i], outputPoint[i], outputNormal[i], &k, minimal_depth, scalingProtectionSphere, leavesapssNodeStats, node, stepMaxSize, nodeInSharedMemory, sharedMemorySize);
#else
			oneStepProjection(outputPoint[i], outputPoint[i], outputNormal[i], kernel, minimal_depth, scalingProtectionSphere, leavesapssNodeStats, node, stepMaxSize);
#endif
		}
	}
}

void Fastapss::project(unsigned int nbOfVectors, glm::vec3 *outputPoints, glm::vec3 *outputNormals, unsigned int n_iterations, const Kernel &kernel, float stepMaxSize) const
{
	if (nbOfVectors == 0)
	{
		std::cerr << "No points to project" << std::endl;
		return;
	}
	int blockSize = 128;
	dim3 numBlocks = computeNbBlocks(nbOfVectors, blockSize);
	if (stepMaxSize < 0)
		stepMaxSize =  m_apssOctree->getBoundingBox().radius();
	projectCuda<<<numBlocks, blockSize>>>(nbOfVectors, m_outPts, m_outPts, m_outNmls, n_iterations, m_kernelCuda, m_minDepth, m_scalingProtectionSphere, *m_apssNodeStatsGPU, m_nodesBisGPU, stepMaxSize, blockSize);
	gpuErrchk( hipPeekAtLastError() );

	if(outputPoints) gpuErrchk( hipMemcpy(outputPoints, m_outPts, nbOfVectors * sizeof (float3), hipMemcpyDeviceToHost));
	if(outputNormals) gpuErrchk( hipMemcpy(outputNormals, m_outNmls, nbOfVectors * sizeof (float3), hipMemcpyDeviceToHost));
}

void Fastapss::copyToGPU(const Kernel &kernel, unsigned int nbOfVectors)
{
	//Allocate memory on GPU
	gpuErrchk( hipPeekAtLastError() );
	std::cout << "Tree is being copied to GPU memory... " << std::flush;
	m_numberOfNodes = m_apssOctree->get_root()->nbOfNodes();
	std::cout << "Number of nodes : " << m_numberOfNodes << std::endl;
	
	//Fill GPU memory
	m_nodesBisCPU = new octreeNodeGPU2[m_numberOfNodes];
	copy(m_apssOctree->get_root(), m_nodesBisCPU, m_numberOfNodes);
	gpuErrchk(hipMalloc(&m_nodesBisGPU, m_numberOfNodes * sizeof(octreeNodeGPU2)));
	gpuErrchk(hipMemcpy(m_nodesBisGPU, m_nodesBisCPU, m_numberOfNodes * sizeof(octreeNodeGPU2), hipMemcpyHostToDevice));

	// Kernel
	KernelGPU kernelOnCPU;
	kernelOnCPU.copy(kernel);
	gpuErrchk(hipMalloc(&m_kernelCuda, sizeof(KernelGPU)));
	gpuErrchk(hipMemcpy(m_kernelCuda, &kernelOnCPU, sizeof(KernelGPU), hipMemcpyHostToDevice));

	// Stats
	m_apssNodeStatsCPU = new apssNodeStatsGPUSoA();
	m_apssNodeStatsCPU->copy(m_apssOctree->getPointsApssNodeStats());
	gpuErrchk(hipMalloc(&m_apssNodeStatsGPU, sizeof(apssNodeStatsGPUSoA)));
	gpuErrchk(hipMemcpy(m_apssNodeStatsGPU, m_apssNodeStatsCPU, sizeof(apssNodeStatsGPUSoA), hipMemcpyHostToDevice));
	std::cout << "    Done" << std::endl;
	std::cout << "Tree depth: " << m_apssOctree->get_root()->computeDepth() << std::endl;
}

void Fastapss::eraseFromGPU()
{
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk(hipFree(m_kernelCuda));

	// Stats
	m_apssNodeStatsCPU->erase();
	delete m_apssNodeStatsCPU;
	m_apssNodeStatsCPU = nullptr;
	gpuErrchk(hipFree(m_apssNodeStatsGPU));
	m_apssNodeStatsGPU = nullptr;

	// Nodes bis, note that the allocation for "indices" is shared, so
	// we shall only free it once. The actual pointer can be found in
	// the first node that has a valid "indices" pointer.
	for (unsigned int i = 0; i < m_numberOfNodes; i++)
		if(m_nodesBisCPU[i].indicesSize > 0)
		{
			gpuErrchk( hipFree(m_nodesBisCPU[i].indices) );
			break;
		}
	delete[] m_nodesBisCPU;
	m_nodesBisCPU = nullptr;
	gpuErrchk(hipFree(m_nodesBisGPU));
	m_nodesBisGPU = nullptr;
}

void Fastapss::projectCPU(const glm::vec3 &qStart, glm::vec3 &outputPoint, glm::vec3 &outputNormal, unsigned int n_iterations, const Kernel &kernel, float stepMaxSize) const
{
	m_apssOctree->projectCPU(qStart, outputPoint, outputNormal, n_iterations, true, kernel, stepMaxSize);
}
