#include "hip/hip_runtime.h"
// --------------------------------------------------------------------------
// This file is part of the reference implementation for the paper
//    Moving Level-of-Detail Surfaces.
//    C. Mercier, T. Lescoat, P. Roussillon, T. Boubekeur, and J-M. Thiery
//    ACM Transaction On Graphics 2022
//    DOI: 10.1145/3528223.3530151
//
// All rights reserved. Use of this source code is governed by a
// MIT license that can be found in the LICENSE file.
// --------------------------------------------------------------------------

#pragma once

#include "GPUUtils.h"

#include <thrust/host_vector.h>
#include <thrust/device_vector.h>
#include <thrust/device_ptr.h>
#include <thrust/scan.h>
#include <thrust/reduce.h>
#include <thrust/fill.h>
#include <thrust/system_error.h>
#include <hip/hip_runtime_api.h>
#include <cuda_gl_interop.h>

//////////////////////////////////////////////////////////////////////////
///
/// General GPU functions
///
//////////////////////////////////////////////////////////////////////////

#define gpuErrchk(ans) { gpuAssert((ans), #ans, __FILE__, __LINE__); }
inline void gpuAssert(hipError_t code, const char* command, const char *file, int line, bool abort=true)
{
	//printf("%s = %i\n", command, (int)code);
	if (code != hipSuccess)
	{
		fprintf(stderr,"GPUassert: %s %s %d\n", hipGetErrorString(code), file, line);
		if (abort) exit(code);
	}
}

//basic maths device functions to use in cuda kernels //////////////////////////////////////////////

typedef struct{
	float q[9];
}mat33;

inline __host__ __device__ void setZero(mat33 &m){
	for (int i = 0; i < 9; i++){
		m.q[i] = 0.0;
	}
}

inline __host__ __device__ mat33 setMat33(float m00, float m01, float m02, float m10, float m11, float m12, float m20, float m21, float m22){
	mat33 res;
	res.q[0] = m00;
	res.q[1] = m01;
	res.q[2] = m02;
	res.q[3] = m10;
	res.q[4] = m11;
	res.q[5] = m12;
	res.q[6] = m20;
	res.q[7] = m21;
	res.q[8] = m22;
	return res;
}

inline __host__ __device__ mat33 identity33(){
	mat33 id = setMat33(1.0, 0.0, 0.0,
						0.0, 1.0, 0.0,
						0.0, 0.0, 1.0);
	return id;
}

inline __host__ __device__ mat33 getTranspose(mat33 m){
	mat33 res;
	res = setMat33(m.q[0], m.q[3], m.q[6], m.q[1], m.q[4], m.q[7], m.q[2], m.q[5], m.q[8]);
	return res;
}

inline __host__ __device__ float3 operator*(mat33 m, float3 p){
	float3 res = make_float3(m.q[0] * p.x + m.q[1] * p.y + m.q[2] * p.z,
			m.q[3] * p.x + m.q[4] * p.y + m.q[5] * p.z,
			m.q[6] * p.x + m.q[7] * p.y + m.q[8] * p.z);
	return res;
}

inline __host__ __device__ mat33 operator*(float f, mat33 m){
	mat33 res = setMat33(f * m.q[0], f * m.q[1], f * m.q[2],
			f * m.q[3], f * m.q[4], f * m.q[5],
			f * m.q[6], f * m.q[7], f * m.q[8]);
	return res;
}

inline __host__ __device__ mat33 operator+(mat33 m1, mat33 m2){
	mat33 res = setMat33(m1.q[0] + m2.q[0], m1.q[1] + m2.q[1], m1.q[2] + m2.q[2],
			m1.q[3] + m2.q[3], m1.q[4] + m2.q[4], m1.q[5] + m2.q[5],
			m1.q[6] + m2.q[6], m1.q[7] + m2.q[7], m1.q[8] + m2.q[8]);
	return res;
}

inline __host__ __device__ mat33 operator*(mat33 m1, mat33 m2){
	mat33 res = setMat33(m1.q[0] * m2.q[0] + m1.q[1] * m2.q[3] + m1.q[2] * m2.q[6],
			m1.q[0] * m2.q[1] + m1.q[1] * m2.q[4] + m1.q[2] * m2.q[7],
			m1.q[0] * m2.q[2] + m1.q[1] * m2.q[5] + m1.q[2] * m2.q[8],
			m1.q[3] * m2.q[0] + m1.q[4] * m2.q[3] + m1.q[5] * m2.q[6],
			m1.q[3] * m2.q[1] + m1.q[4] * m2.q[4] + m1.q[5] * m2.q[7],
			m1.q[3] * m2.q[2] + m1.q[4] * m2.q[5] + m1.q[5] * m2.q[8],
			m1.q[6] * m2.q[0] + m1.q[7] * m2.q[3] + m1.q[8] * m2.q[6],
			m1.q[6] * m2.q[1] + m1.q[7] * m2.q[4] + m1.q[8] * m2.q[7],
			m1.q[6] * m2.q[2] + m1.q[7] * m2.q[5] + m1.q[8] * m2.q[8]);
	return res;
}

inline __host__ __device__ float3 operator-( float3 a){
	return make_float3(-a.x, -a.y, -a.z);
}

inline __host__ __device__ float3 operator-( float3 a, float3 b){
	return make_float3(a.x-b.x, a.y-b.y, a.z-b.z);
}

inline __host__ __device__ double3 operator-( double3 a, double3 b){
	return make_double3(a.x-b.x, a.y-b.y, a.z-b.z);
}

inline __host__ __device__ float3 operator+( float3 a, float3 b){
	return make_float3(a.x+b.x, a.y+b.y, a.z+b.z);
}

inline __host__ __device__ double3 operator+( double3 a, double3 b){
	return make_double3(a.x+b.x, a.y+b.y, a.z+b.z);
}

inline __host__ __device__ float3 operator*( float3 a, float3 b){
	return make_float3(a.x*b.x, a.y*b.y, a.z*b.z);
}

inline __host__ __device__ float3 operator/( float3 a, float b){
	return make_float3(a.x/b, a.y/b, a.z/b);
}

inline __host__ __device__ double3 operator/( double3 a, double b){
	return make_double3(a.x/b, a.y/b, a.z/b);
}

inline __host__ __device__ float3 operator*( float a, float3 b){
	return make_float3(a*b.x, a*b.y, a*b.z);
}


inline __device__ double3 cross(double3 a, double3 b){
	return make_double3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

inline __device__ double dot(double3 a, double3 b){
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

inline __device__ double norme(double3 x){
	return sqrt(x.x * x.x + x.y * x.y + x.z * x.z);
}

inline __device__ double squaredNorm(double3 x){
	return (x.x * x.x + x.y * x.y + x.z * x.z);
}

inline __device__ float3 cross(float3 a, float3 b){
	return make_float3(a.y * b.z - a.z * b.y, a.z * b.x - a.x * b.z, a.x * b.y - a.y * b.x);
}

inline __device__ float dot(float3 a, float3 b){
	return a.x * b.x + a.y * b.y + a.z * b.z;
}

inline __device__ float norme(float3 const & a){
	return __fsqrt_rn(a.x * a.x + a.y * a.y + a.z * a.z);
}

inline __device__ float squaredNorm(float3 x){
	return (x.x * x.x + x.y * x.y + x.z * x.z);
}

inline __device__ float squaredDist(float3 x1, float3 x2){
	return ((x1.x - x2.x) * (x1.x - x2.x) + (x1.y - x2.y) * (x1.y - x2.y) + (x1.z - x2.z) * (x1.z - x2.z));
}

inline __device__ float dist(float3 x1, float3 x2){
	return __fsqrt_rn(squaredDist(x1, x2));
}

inline __device__ float3 normalize(float3 x){
	float norm = __fsqrt_rn(x.x * x.x + x.y * x.y + x.z * x.z);
	if(norm!=0){
		return make_float3(x.x / norm, x.y / norm, x.z / norm);
	}else{
		return make_float3(0.0, 0.0, 0.0);
	}
}

inline __device__ double3 normalize(double3 x){
	double norm = sqrt(x.x * x.x + x.y * x.y + x.z * x.z);
	if(norm!=0){
		return make_double3(x.x / norm, x.y / norm, x.z / norm);
	}else{
		return make_double3(0.0, 0.0, 0.0);
	}
}

inline float3 glmToFloat3(glm::vec3 a)
{
	return make_float3(a.x, a.y, a.z);
}

//End of basic math

inline dim3 computeNbBlocks(unsigned int nbThreads, unsigned int nbThreadsPerBlock){
	dim3 nbBlocks = ceil((float)(nbThreads)/(float)nbThreadsPerBlock);
	if (nbBlocks.x > 65535){
		nbBlocks.y = ceil((float)nbBlocks.x / (float)65535);
		nbBlocks.x = 65535;
	}
	return nbBlocks;
}

//////////////////////////////////////////////////////////////////////////
///
/// GPU Kernel
///
//////////////////////////////////////////////////////////////////////////

struct KernelGPU{
	enum KernelType { GAUSSIAN , WENDLAND , SINGULAR , GAUSSIAN_MULTIPLE };
	float sigma;
	float exponent;
	float constant_shift;
	float *sigmas;
	unsigned int sigmasSize;

	KernelType type;

	__device__ float w( float3 const & p , float3 const & eta) const {
		if( type == KernelGPU::GAUSSIAN_MULTIPLE ) {
			float sum_g = 0.f;//constant_shift;
			for( unsigned int i = 0 ; i < sigmasSize ; ++i )
				sum_g = sum_g + __fdiv_rn( __expf( __fdiv_rn( - squaredNorm(p-eta) , (2.f * sigmas[i]*sigmas[i]) ) ) , (sigmas[i]*sigmas[i]*sigmas[i]) )  ;
			return sum_g;
		}
		if( type == KernelGPU::SINGULAR )
			return __fdiv_rn(__powf(max(constant_shift, 0.0001), exponent), __powf(__powf(norme(p - eta), 2) + constant_shift, exponent));
		if( type == KernelGPU::GAUSSIAN )
		{
			return __expf( __fdiv_rn(- squaredNorm(p-eta) , (2.0 * sigma*sigma)) ) + constant_shift;
		}
		return 1.f;
	}
	__host__ void copy(Kernel const & a)
	{
		if(a.type == Kernel::GAUSSIAN)
			type = KernelGPU::GAUSSIAN;
		if(a.type == Kernel::GAUSSIAN_MULTIPLE)
			type = KernelGPU::GAUSSIAN_MULTIPLE;
		if(a.type == Kernel::SINGULAR)
			type = KernelGPU::SINGULAR;
		sigma = a.sigma;
		exponent = a.exponent;
		constant_shift = a.constant_shift;
		sigmasSize = a.sigmas.size();
		if (sigmasSize !=0)
		{
			gpuErrchk( hipMalloc(&sigmas, a.sigmas.size() * sizeof(float)) );
			gpuErrchk( hipMemcpy(sigmas, a.sigmas.data(), a.sigmas.size() * sizeof(float), hipMemcpyHostToDevice) );
		}
	}
	__host__ void print() const
	{
		std::cout << "sigma: " << sigma << std::endl;
		std::cout << "exponent: " << exponent << std::endl;
		std::cout << "constant_shift: " << constant_shift << std::endl;
		std::cout << "sigmasSize: " << sigmasSize << std::endl;
		for (unsigned i=0; i<sigmasSize; i++)
			std::cout << i << " " << sigmas[i] << std::endl;
	}
};

struct BBOXGPU
{
	float3 bb,BB;

	__device__ inline void squareDiagonal(float & sd) const { sd = squaredNorm(BB - bb); }
	__device__ inline void diagonal(float & d) const { d = sqrt( (float)squaredNorm(BB - bb) ); }
	__device__ inline void radius(float & r) const {float d; diagonal(d); r = d / 2.0; }
	__device__ inline void center(float3 & c) const { c = (bb + BB) / 2.0; }
	__device__ inline void squareRadius(float & sr) const { float sd; squareDiagonal(sd); sr = sd / 4.0; }
	__host__ void copy(BBOX const & a)
	{
		bb = make_float3(a.bb.x, a.bb.y, a.bb.z);
		BB = make_float3(a.BB.x, a.BB.y, a.BB.z);
	}
};

struct octreeNodeGPU2 {
	unsigned int depth;
	float3 center;
	float radius;
	float3 s_ai_pi , s_ai_ni;
	float s_ai_pi_ni , s_ai_pi_pi;
	float s_ai;
	unsigned int indicesSize;
	unsigned int* indices = nullptr;
	unsigned int numberOfChildren;
	int firstChild;
	int nextBrother;
	int father;
};

struct apssStatsGPU{
	float3 s_ai_wi_pi , s_ai_wi_ni;
	float s_ai_wi, s_ai_wi_pi_ni , s_ai_wi_pi_pi;

	__device__ void init()
	{
		s_ai_wi = 0;
		s_ai_wi_ni = make_float3(0.0, 0.0, 0.0);
		s_ai_wi_pi = make_float3(0.0, 0.0, 0.0);
		s_ai_wi_pi_ni = 0;
		s_ai_wi_pi_pi = 0;
	}

	__device__ void init(float w, octreeNodeGPU2 * const node)
	{
		s_ai_wi = w * node->s_ai;
		s_ai_wi_ni = w * node->s_ai_ni;
		s_ai_wi_pi = w * node->s_ai_pi;
		s_ai_wi_pi_ni = w * node->s_ai_pi_ni;
		s_ai_wi_pi_pi = w * node->s_ai_pi_pi;
	}

	__device__ void init(float w, octreeNodeGPU2 const node)
	{
		s_ai_wi = w * node.s_ai;
		s_ai_wi_ni = w * node.s_ai_ni;
		s_ai_wi_pi = w * node.s_ai_pi;
		s_ai_wi_pi_ni = w * node.s_ai_pi_ni;
		s_ai_wi_pi_pi = w * node.s_ai_pi_pi;
	}

	__device__ void operator += (apssStatsGPU const * o) {
		s_ai_wi_pi = s_ai_wi_pi + o->s_ai_wi_pi;
		s_ai_wi_ni = s_ai_wi_ni + o->s_ai_wi_ni;
		s_ai_wi += o->s_ai_wi;
		s_ai_wi_pi_ni += o->s_ai_wi_pi_ni;
		s_ai_wi_pi_pi += o->s_ai_wi_pi_pi;
	}
	__device__ void add (apssStatsGPU const *i1){
		this->s_ai_wi_pi = i1->s_ai_wi_pi + s_ai_wi_pi;
		this->s_ai_wi_ni = i1->s_ai_wi_ni + s_ai_wi_ni;
		this->s_ai_wi = i1->s_ai_wi + s_ai_wi;
		this->s_ai_wi_pi_ni = i1->s_ai_wi_pi_ni + s_ai_wi_pi_ni;
		this->s_ai_wi_pi_pi = i1->s_ai_wi_pi_pi + s_ai_wi_pi_pi;
	}
	__device__ void add (apssStatsGPU const &i1){
		s_ai_wi_pi = i1.s_ai_wi_pi + s_ai_wi_pi;
		s_ai_wi_ni = i1.s_ai_wi_ni + s_ai_wi_ni;
		s_ai_wi = i1.s_ai_wi + s_ai_wi;
		s_ai_wi_pi_ni = i1.s_ai_wi_pi_ni + s_ai_wi_pi_ni;
		s_ai_wi_pi_pi = i1.s_ai_wi_pi_pi + s_ai_wi_pi_pi;
	}
	__device__ void times (apssStatsGPU * const i, float const f){
		this->s_ai_wi_pi = f * i->s_ai_wi_pi;
		this->s_ai_wi_ni = f * i->s_ai_wi_ni;
		this->s_ai_wi = i->s_ai_wi * f;
		this->s_ai_wi_pi_ni = i->s_ai_wi_pi_ni * f;
		this->s_ai_wi_pi_pi = i->s_ai_wi_pi_pi * f;
	}
	__device__ void times (apssStatsGPU const & i, float const f){
		s_ai_wi_pi = f * i.s_ai_wi_pi;
		s_ai_wi_ni = f * i.s_ai_wi_ni;
		s_ai_wi = i.s_ai_wi * f;
		s_ai_wi_pi_ni = i.s_ai_wi_pi_ni * f;
		s_ai_wi_pi_pi = i.s_ai_wi_pi_pi * f;
	}
	__host__ void copy(apssStats const & a)
	{
		s_ai_wi = a.s_ai_wi;
		s_ai_wi_ni = make_float3(a.s_ai_wi_ni.x, a.s_ai_wi_ni.y, a.s_ai_wi_ni.z);
		s_ai_wi_pi = make_float3(a.s_ai_wi_pi.x, a.s_ai_wi_pi.y, a.s_ai_wi_pi.z);
		s_ai_wi_pi_ni = a.s_ai_wi_pi_ni;
		s_ai_wi_pi_pi = a.s_ai_wi_pi_pi;
	}
};

void APSS::stop()
{
	gpuErrchk( hipProfilerStop() );
}

void APSS::eraseAndRestart()
{
	eraseFromGPU();
	delete(m_apssOctree);
	m_apssOctree = new APSSOctree();
}

void APSS::copyPointsToGPU(unsigned int nbOfVectors, glm::vec3 * outputPoints)
{
	gpuErrchk( hipMalloc(&m_outPts,  nbOfVectors * sizeof(float3)));
	gpuErrchk( hipMalloc(&m_outNmls, nbOfVectors * sizeof(float3)));
	gpuErrchk( hipMemcpy(m_outPts, outputPoints, nbOfVectors * sizeof (float3), hipMemcpyHostToDevice));
}

__global__
void reorganizePts(unsigned n, unsigned const * invalidPts, unsigned endValid, float3 * outputPoint)
{
	int index = threadIdx.x + blockIdx.x * blockDim.x;
	int stride = blockDim.x * gridDim.x;
	for (int i = index; i < n; i+=stride)
	{
		float3 temp = outputPoint[invalidPts[i]];
		outputPoint[invalidPts[i]] = outputPoint[endValid - 1 - i];
		outputPoint[endValid - 1 - i] = temp;
	}
}

void APSS::printApss() const
{
	std::cout << "m_numberOfNodes: " << m_numberOfNodes << std::endl;
	std::cout << "m_scalingProtectionSphere: " << m_scalingProtectionSphere << std::endl;
	std::cout << "m_minDepth: " << m_minDepth << std::endl;
	std::cout << "KernelCuda:" << std::endl;
	m_kernelCuda->print();
	m_apssOctree->printState();
}

void APSS::reorganizePoints(const std::vector<unsigned>* invalidPts, unsigned endValid)
{
	gpuErrchk( hipPeekAtLastError() );
	unsigned nbOfElmts = invalidPts->size();
	unsigned * invalid = NULL;
	gpuErrchk( hipMalloc(&invalid, nbOfElmts * sizeof(unsigned)));
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk( hipMemcpy(invalid, invalidPts->data(), nbOfElmts * sizeof (int), hipMemcpyHostToDevice));
	gpuErrchk( hipPeekAtLastError() );
	int blockSize = 128;
	dim3 numBlocks = computeNbBlocks(nbOfElmts, blockSize);
	reorganizePts<<<numBlocks, blockSize>>>(nbOfElmts, invalid, endValid, m_outPts);
	gpuErrchk( hipPeekAtLastError() );
	gpuErrchk(hipFree(invalid));
}

void APSS::erasePointsFromGPU()
{
	if (m_outPts)
	{
		gpuErrchk(hipFree(m_outPts));
		m_outPts = NULL;
	}
	if (m_outNmls)
	{
		gpuErrchk(hipFree(m_outNmls));
		m_outNmls = NULL;
	}
}

void APSS::updateKernel(Kernel const & kernel)
{
	KernelGPU kernelOnCPU;
	kernelOnCPU.copy(kernel);
	gpuErrchk(hipMalloc(&m_kernelCuda, sizeof(KernelGPU)));
	gpuErrchk(hipMemcpy(m_kernelCuda, &kernelOnCPU, sizeof(KernelGPU), hipMemcpyHostToDevice));
}
